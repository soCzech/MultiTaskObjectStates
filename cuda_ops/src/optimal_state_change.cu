#include "hip/hip_runtime.h"
#include "common.hpp"

__global__ void SingleStateChangeKernel(
    const float* state_tensor,
    const float* action_tensor,
    const int* lens,
    int* state_targets,
    int* action_targets,
    const int delta,
    const int kappa,
    const int max_action_state_distance
) {
    const int batch_idx = blockIdx.x;
    const int video_len = blockDim.x;
    const int state1_pos = threadIdx.x;
    const int actual_len = lens[batch_idx];

    // get pointer to shared memory
    extern __shared__ char shared_mem[];
    int* state1_to_action_pos = reinterpret_cast<int*>(shared_mem);
    int* state1_to_state2_pos = state1_to_action_pos + video_len;
    float* state1_to_score = reinterpret_cast<float*>(state1_to_state2_pos + video_len);
    float* action_tensor_shared = state1_to_score + video_len;
    float* state_tensor_shared = action_tensor_shared + video_len;

    // load action and state tensors into shared memory
    action_tensor_shared[state1_pos] = action_tensor[batch_idx * video_len + state1_pos];
    state_tensor_shared[2 * state1_pos + 0] = state_tensor[batch_idx * video_len * 2 + state1_pos * 2 + 0];
    state_tensor_shared[2 * state1_pos + 1] = state_tensor[batch_idx * video_len * 2 + state1_pos * 2 + 1];

    __syncthreads();

    float best_score = -std::numeric_limits<float>::infinity();
    int best_action_pos = 0, best_state2_pos = 0; // position of states/action for videos shorter than 3

    for (int action_pos = state1_pos + 1; action_pos <= state1_pos + max_action_state_distance && action_pos < actual_len - 1; ++action_pos) { // -1: need at least one position for state2
        float action_score = action_tensor_shared[action_pos];

        for (int state2_pos = action_pos + 1; state2_pos <= action_pos + max_action_state_distance && state2_pos < actual_len; ++state2_pos) {
            float state2_score = state_tensor_shared[2 * state2_pos + 1]; // 2 states, +1 for second state

            float score = action_score * state2_score;
            if (score > best_score) {
                best_score = score;
                best_action_pos = action_pos;
                best_state2_pos = state2_pos;
            }
        }
    }

    state1_to_action_pos[state1_pos] = best_action_pos;
    state1_to_state2_pos[state1_pos] = best_state2_pos;
    state1_to_score[state1_pos] = best_score * state_tensor_shared[2 * state1_pos + 0];

    __syncthreads();

    if (state1_pos == 0) { // compute reduction only on the first thread
        best_score = state1_to_score[0];
        int best_state1_pos = 0;
        for (int i = 1; i < actual_len - 2; ++i) { // -2: need at least one position for action and one for state2
            if (best_score < state1_to_score[i]) {
                best_state1_pos = i;
                best_score = state1_to_score[i];
            }
        }
        best_action_pos = state1_to_action_pos[best_state1_pos];
        best_state2_pos = state1_to_state2_pos[best_state1_pos];

        // FILL state_targets TENSOR
        // 0 .. default - no label
        // 1 .. initial state label
        // 2 .. end state label
        for (int i = best_state1_pos - delta; i <= best_state1_pos + delta; ++i) {
            if (i < 0 || i >= actual_len) continue;
            state_targets[batch_idx * video_len + i] = 1;
        }
        for (int i = best_state2_pos - delta; i <= best_state2_pos + delta; ++i) {
            if (i < 0 || i >= actual_len) continue;
            state_targets[batch_idx * video_len + i] = 2;
        }

        // FILL action_targets TENSOR
        // 0 .. default - no label
        // 1 .. no-action label
        // 2 .. action label
        for (int i = 0; i <= delta; ++i) {
            int j = best_action_pos - i - kappa;
            if (j < 0) {
                action_targets[batch_idx * video_len + 0] = 1;
            } else {
                action_targets[batch_idx * video_len + j] = 1;
            }

            int k = best_action_pos + i + kappa;
            if (k >= actual_len) {
                action_targets[batch_idx * video_len + actual_len - 1] = 1;
            } else {
                action_targets[batch_idx * video_len + k] = 1;
            }
        }
        for (int i = best_action_pos - delta; i <= best_action_pos + delta; ++i) {
            if (i < 0 || i >= actual_len) continue;
            action_targets[batch_idx * video_len + i] = 2;
        }
    }
}

std::vector<torch::Tensor> optimal_state_change(
    torch::Tensor state_tensor, torch::Tensor action_tensor, torch::Tensor lens, int delta, int kappa, int max_action_state_distance) {

    CHECK_CUDA_INPUT(state_tensor);
    CHECK_CUDA_INPUT(action_tensor);
    CHECK_CUDA_INPUT(lens);

    int batch_size = state_tensor.size(0);
    int video_len = state_tensor.size(1);

    TORCH_CHECK(state_tensor.size(2) == 2, "state_tensor must be of shape [batch, video_len, 2]")
    TORCH_CHECK(action_tensor.size(2) == 1, "action_tensor must be of shape [batch, video_len, 1]")

    auto options = torch::TensorOptions().dtype(torch::kInt).device(torch::kCUDA);
    auto state_targets = torch::zeros({batch_size, video_len}, options);
    auto action_targets = torch::zeros({batch_size, video_len}, options);

    const int threads = video_len;
    const int blocks = batch_size;
    // store in shared memory:
    //  best action position for each state1 position (1x int)
    //  best state2 position for each state1 position (1x int)
    //  best score for each state1 position (1x float)
    //  action tensor (1x float)
    //  state tensor (2x float)
    const int shared_mem = video_len * (2 * sizeof(int) + 4 * sizeof(float));
    SingleStateChangeKernel<<<blocks, threads, shared_mem>>>(
            state_tensor.data_ptr<float>(),
            action_tensor.data_ptr<float>(),
            lens.data_ptr<int>(),
            state_targets.data_ptr<int>(),
            action_targets.data_ptr<int>(),
            delta,
            kappa,
            max_action_state_distance);

    return std::vector<torch::Tensor>{state_targets, action_targets};
}
